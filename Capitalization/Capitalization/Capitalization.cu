#include <hip/hip_runtime.h>

#include <iostream>


#define N 32 //allocate space for vars; this will end up being the number of blocks to iterate over (we want this to be multiples of 32)

__global__ void Caps(char *c, int *b)
{
	int tid = blockIdx.x;
	if (tid < N)
	{
		if (b[tid] == 1)
		{
			int ascii = (int)c[tid];
			ascii -= 32;
			c[tid] = (char)ascii;
		}
	}
	
}

int main()
{
	int a[] = {1, 4, 6, 8, 11, 30};
	char String[N];
	int *b;
	char *c;
	
	//geneate 32 character string
	for(int i=0;i<N;i++){
		if (i % 5 == 0) { String[i] = 'a'; }
		if (i % 5 == 1) { String[i] = 'b'; }
		if (i % 5 == 2) { String[i] = 'c'; }
		if (i % 5 == 3) { String[i] = 'd'; }
		if (i % 5 == 4) { String[i] = 'e'; }
	}

	int temp[sizeof(String)/sizeof(char)];
	for (int i = 0; i < (sizeof(String)/sizeof(char)); i++)
	{
		temp[i]=0;
	}
	for (int i = 0; i < (sizeof(a)/sizeof(int)); i++)
	{
		int val=a[i];
		temp[val]=1;
	}

	hipMalloc((void**)&c, N * sizeof(char));
	hipMalloc((void**)&b, N * sizeof(int));
	hipMemcpy(b, &temp, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(c, &String, N * sizeof(char), hipMemcpyHostToDevice);

	Caps<<<N, 1>>>(c, b);

	hipMemcpy(&String, c, N * sizeof(char), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
	{
		printf("%c", String[i]);
	}
	printf("\n");

	hipFree(b);
	hipFree(c);

	return 0;
}