#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err)(HandleError(err, __FILE__, __LINE__))

//star character is a pointer
__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int main(void)
{
	/*int c;
	int *dev_c; //pointer declaration
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	add<<<1, 1 >>>(2, 7, dev_c);

	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));*/

	int count;
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int x = 0; x < count; x++)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop, x));
		printf("Device %d \n", x);
		printf("Name: %s\n", prop.name);
		printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		if (prop.deviceOverlap) { printf("Overlap Enabled\n"); };
		if (prop.kernelExecTimeoutEnabled) { printf("Kernal Timeout Enabled\n"); };
		printf("Memory Info\n");
		printf("Total global mem: %ld\n", prop.totalGlobalMem);	
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf("Multiprocessor info\n");
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp %d\n", prop.sharedMemPerBlock);
		printf("Registers per mp %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Memory per block: %d\n", prop.sharedMemPerBlock);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	}
	//page 34 - 55 in pdf
	//printf("%d",count);
	//printf("2 + 7 = %d\n", c);
	//hipFree(dev_c);
	getchar();
	return 0;
}