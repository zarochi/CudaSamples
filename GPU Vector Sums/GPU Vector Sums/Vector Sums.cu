#include <iostream>
#include <hip/hip_runtime.h>



#define N 10

__global__ void add(int *a, int *b, int *c)
{
	//blockIdx is the value of the block index for whichever block is running the code
	int tid = blockIdx.x;//handle the data at this index
	//blockIdx has 2 dimensions; x and y. We only need one dimension
	if(tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(void)
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	//GPU memory allocation
	//cudaMalloc((return type)&pointer, size in memory);
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	//fill arrays a and b on the CPU with arbitrary values
	for (int x = 0; x < N; x++)
	{
		a[x] = -x;
		b[x] = x*x;
	}

	//Copy arrays to GPU
	//cudaMemcpy(to, from, size in memory, direction);
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	//N - the number of parallel blocks in which we would like the device to execute the kernel
	add<<<N, 1 >>>(dev_a, dev_b, dev_c);

	//Copy array c back from the GPU
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	getchar();
	return 0;
	//page 59 - 80 in pdf
}