#include <hip/hip_runtime.h>


#include <iostream>

__global__ void ThirdAngle(int *a1, int *a2, int *a3)
{
	*a3 = (180-*a1-*a2);
}

int main()
{
	int *a3, *a2, *a1;
	int var1, var2, angle3;

	printf("Enter angle1\n");
	scanf("%d", &var1);
	printf("Enter angle2\n");
	scanf("%d", &var2);

	//Cuda goodness
	hipMalloc((void**)&a3, sizeof(int));
	hipMalloc((void**)&a2, sizeof(int));
	hipMalloc((void**)&a1, sizeof(int));
	//Copy read vars to cuda vars
	hipMemcpy(a1, &var1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a2, &var2, sizeof(int), hipMemcpyHostToDevice);
	ThirdAngle<<<1,1>>>(a1, a2, a3);//Run Cuda function on single block
	hipMemcpy(&angle3, a3, sizeof(int), hipMemcpyDeviceToHost);//Nab the angle back to angle3

	printf("Third Angle:\n %d", angle3);
	getchar();
	return 0;
}