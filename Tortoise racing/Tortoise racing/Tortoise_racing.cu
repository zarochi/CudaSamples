#include <hip/hip_runtime.h>

#include <iostream>


#define N 32

__global__ void Turtle(int *v1, int *v2, int *lead)
{
	if (*v1 >= *v2)
	{
		printf("%d, %d, %d", -1, -1,-1);
	}
	else
	{
		printf("%d, %d, %d\n", *v1, *v2, *lead);
		double _result = ((float)*lead)/(((float)*v2)-((float)*v1));
		int h = _result;
		int m = _result * 60 - h*60;
		int s = (_result * 3600) -m*60;
		printf("%.3f\n",_result);
		printf("%d, %d, %d\n", h, m, s);
		
	}
}

int main()
{
	int _v1 = 720;
	int _v2 = 850;
	int _lead = 70;
	int *v1, *v2, *lead;

	hipMalloc((void**)&v1, sizeof(int));
	hipMalloc((void**)&v2, sizeof(int));
	hipMalloc((void**)&lead, sizeof(int));

	hipMemcpy(v1, &_v1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(v2, &_v2, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(lead, &_lead, sizeof(int), hipMemcpyHostToDevice);

	Turtle<<<1, 1>>>(v1, v2, lead);

	hipFree(v1);
	hipFree(v2);
	hipFree(lead);

	return 0;
}

//lead 70 feet
//how long to catch A
//hour min and sec return value
//-1, -1, -1 if v1>=v2