#include <hip/hip_runtime.h>

#include <iostream>


__global__ void W4W(int *w, int *out)
{
	int tid = blockIdx.x;
	int weight[sizeof(w)/sizeof(int)];
		char c1 = (w[tid]/100)+48;
		char c2 = ((w[tid]%100)/10)+48;
		char c3 = w[tid]%10+48;
		weight[tid]=(w[tid]/100)+ ((w[tid] % 100) / 10) + w[tid] % 10;
		printf("%d, %c, %c, %c, %d\n", w[tid], c1, c2, c3, weight[tid]);
		if (tid != 0)
		{
			if (weight[tid - 1] > weight[tid])
			{
				int x,y;
				x=w[tid -1];y=w[tid];//collapsed to reserve pixels
				out[tid -1]=y;out[tid]=x;
				x=0;y=0;
				x=weight[tid -1];y=weight[tid];
				weight[tid -1]=y;weight[tid]=x;
			}
			if (weight[tid - 1] = weight[tid])
			{

			}
		}
}

int main()
{
	//weight is the sum of the numbers
	//like numbers differentiated by string
	const int size = 9;
	int weights[size] = {56, 65, 74, 100, 99, 68, 86, 180, 90};
	int *out, *w, output[size];
	hipMalloc((void**)&w, size * sizeof(int));
	hipMalloc((void**)&out, size * sizeof(int));
	hipMemcpy(w, &weights, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(out, &output, size * sizeof(int), hipMemcpyHostToDevice);
	W4W<<<size,1>>>(w, out);
	hipMemcpy(&output, out, size * sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<size;i++){printf("%d\n",output[i]); }
	hipFree(out);
	hipFree(w);
	return 0;
}